#include "hip/hip_runtime.h"
#include "cugemm.h"
#include "hip/hip_runtime.h"
#include ""
#include <cassert>

#define d_A(i,j) d_A[k*(i)+(j)]
#define d_B(i,j) d_B[n*(i)+(j)]
#define d_C(i,j) d_C[n*(i)+(j)]

#ifndef KERNEL_SIZE
#define KERNEL_SIZE 8
#endif

__global__ void sgemm_fast_kernel_naive(int k, int m, int n,
    float* d_A, float* d_B, float* d_C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    float c = 0;

    for (int p = 0; p < k; p++)
    {
        c += d_A(i, p) * d_B(p, j);
    }
    d_C(i, j) += c;
}

float sgemm_fast(int k, int m, int n,
	float* A, int lda,
	float* B, int ldb,
	float* C, int ldc)
{
    float* d_A, * d_B, * d_C;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&d_A, sizeof(float) * m * k);
    hipMalloc(&d_B, sizeof(float) * k * n);
    hipMalloc(&d_C, sizeof(float) * m * n);
    hipMemcpy(d_A, A, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * k * n, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, sizeof(float) * m * n);

    dim3 dim_block((m - 1) / (KERNEL_SIZE) + 1, (n - 1) / (KERNEL_SIZE) + 1, 1),
        dim_thread(KERNEL_SIZE, KERNEL_SIZE, 1);

    hipEventRecord(start, 0);
    sgemm_fast_kernel_naive << <dim_block, dim_thread >> > (k, m, n,
        d_A, d_B, d_C);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(C, d_C, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return elapsedTime;
}
