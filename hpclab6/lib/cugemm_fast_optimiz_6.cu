#include "hip/hip_runtime.h"
#include "cugemm.h"
#include "hip/hip_runtime.h"
#include ""
#include <cassert>

#define d_A(i,j) d_A[k*(i)+(j)]
#define d_B(i,j) d_B[n*(i)+(j)]
#define d_C(i,j) d_C[n*(i)+(j)]

#ifndef KERNEL_SIZE
#define KERNEL_SIZE 16
#endif

#define REG_TILE_SIZE 8

__global__ void sgemm_fast_kernel_optimiz_6(int k, int m, int n,
    float* d_A, float* d_B, float* d_C)
{
    __shared__ float sm_A[KERNEL_SIZE][KERNEL_SIZE * REG_TILE_SIZE],
        sm_B[KERNEL_SIZE][KERNEL_SIZE * REG_TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    //Block所计算的kernel的左上角第一个元素在矩阵C中的位置为(Bi,Bj)
    int Bi = KERNEL_SIZE * REG_TILE_SIZE * blockIdx.x;
    int Bj = KERNEL_SIZE * REG_TILE_SIZE * blockIdx.y;

    //线程所计算的micro kernel的左上角第一个元素在矩阵C中的位置为(Ci, Cj)
    int Ci = KERNEL_SIZE * REG_TILE_SIZE * blockIdx.x + threadIdx.x * REG_TILE_SIZE;
    int Cj = KERNEL_SIZE * REG_TILE_SIZE * blockIdx.y + threadIdx.y * REG_TILE_SIZE;

    float4 vec_c0_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c0_47 = make_float4(0.0, 0.0, 0.0, 0.0);
    float4 vec_c1_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c1_47 = make_float4(0.0, 0.0, 0.0, 0.0);
    float4 vec_c2_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c2_47 = make_float4(0.0, 0.0, 0.0, 0.0);
    float4 vec_c3_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c3_47 = make_float4(0.0, 0.0, 0.0, 0.0);
    float4 vec_c4_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c4_47 = make_float4(0.0, 0.0, 0.0, 0.0);
    float4 vec_c5_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c5_47 = make_float4(0.0, 0.0, 0.0, 0.0);
    float4 vec_c6_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c6_47 = make_float4(0.0, 0.0, 0.0, 0.0);
    float4 vec_c7_03 = make_float4(0.0, 0.0, 0.0, 0.0), vec_c7_47 = make_float4(0.0, 0.0, 0.0, 0.0);

    float reg_a0, reg_a1, reg_a2, reg_a3, reg_a4, reg_a5, reg_a6, reg_a7;
    float4 vec_b0_3, vec_b4_7;

    for (int po = 0; po < k; po += KERNEL_SIZE)
    {
        float4 vec_gm_a0 = *reinterpret_cast<float4*>(&d_A(Bi + (tx % REG_TILE_SIZE) * KERNEL_SIZE + ty, po + (tx / REG_TILE_SIZE) * REG_TILE_SIZE + 0));
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 0][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a0.x;
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 1][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a0.y;
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 2][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a0.z;
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 3][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a0.w;
        float4 vec_gm_a1 = *reinterpret_cast<float4*>(&d_A(Bi + (tx % REG_TILE_SIZE) * KERNEL_SIZE + ty, po + (tx / REG_TILE_SIZE) * REG_TILE_SIZE + 4));
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 4][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a1.x;
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 5][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a1.y;
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 6][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a1.z;
        sm_A[(tx / REG_TILE_SIZE) * REG_TILE_SIZE + 7][(tx % REG_TILE_SIZE) * KERNEL_SIZE + ty] = vec_gm_a1.w;

        *reinterpret_cast<float4*>(&sm_B[tx][ty * REG_TILE_SIZE + 0]) = *reinterpret_cast<float4*>(&d_B(po + tx, Cj + 0));
        *reinterpret_cast<float4*>(&sm_B[tx][ty * REG_TILE_SIZE + 4]) = *reinterpret_cast<float4*>(&d_B(po + tx, Cj + 4));

        __syncthreads();
        for (int pi = 0; pi < KERNEL_SIZE; pi++)
        {
            reg_a0 = sm_A[pi][tx * REG_TILE_SIZE + 0];
            reg_a1 = sm_A[pi][tx * REG_TILE_SIZE + 1];
            reg_a2 = sm_A[pi][tx * REG_TILE_SIZE + 2];
            reg_a3 = sm_A[pi][tx * REG_TILE_SIZE + 3];
            reg_a4 = sm_A[pi][tx * REG_TILE_SIZE + 4];
            reg_a5 = sm_A[pi][tx * REG_TILE_SIZE + 5];
            reg_a6 = sm_A[pi][tx * REG_TILE_SIZE + 6];
            reg_a7 = sm_A[pi][tx * REG_TILE_SIZE + 7];

            vec_b0_3 = *reinterpret_cast<float4*>(&sm_B[pi][ty * REG_TILE_SIZE + 0]);
            vec_b4_7 = *reinterpret_cast<float4*>(&sm_B[pi][ty * REG_TILE_SIZE + 4]);

            vec_c0_03.x += reg_a0 * vec_b0_3.x; vec_c0_03.y += reg_a0 * vec_b0_3.y; vec_c0_03.z += reg_a0 * vec_b0_3.z; vec_c0_03.w += reg_a0 * vec_b0_3.w;
            vec_c0_47.x += reg_a0 * vec_b4_7.x; vec_c0_47.y += reg_a0 * vec_b4_7.y; vec_c0_47.z += reg_a0 * vec_b4_7.z; vec_c0_47.w += reg_a0 * vec_b4_7.w;

            vec_c1_03.x += reg_a1 * vec_b0_3.x; vec_c1_03.y += reg_a1 * vec_b0_3.y; vec_c1_03.z += reg_a1 * vec_b0_3.z; vec_c1_03.w += reg_a1 * vec_b0_3.w;
            vec_c1_47.x += reg_a1 * vec_b4_7.x; vec_c1_47.y += reg_a1 * vec_b4_7.y; vec_c1_47.z += reg_a1 * vec_b4_7.z; vec_c1_47.w += reg_a1 * vec_b4_7.w;

            vec_c2_03.x += reg_a2 * vec_b0_3.x; vec_c2_03.y += reg_a2 * vec_b0_3.y; vec_c2_03.z += reg_a2 * vec_b0_3.z; vec_c2_03.w += reg_a2 * vec_b0_3.w;
            vec_c2_47.x += reg_a2 * vec_b4_7.x; vec_c2_47.y += reg_a2 * vec_b4_7.y; vec_c2_47.z += reg_a2 * vec_b4_7.z; vec_c2_47.w += reg_a2 * vec_b4_7.w;

            vec_c3_03.x += reg_a3 * vec_b0_3.x; vec_c3_03.y += reg_a3 * vec_b0_3.y; vec_c3_03.z += reg_a3 * vec_b0_3.z; vec_c3_03.w += reg_a3 * vec_b0_3.w;
            vec_c3_47.x += reg_a3 * vec_b4_7.x; vec_c3_47.y += reg_a3 * vec_b4_7.y; vec_c3_47.z += reg_a3 * vec_b4_7.z; vec_c3_47.w += reg_a3 * vec_b4_7.w;

            vec_c4_03.x += reg_a4 * vec_b0_3.x; vec_c4_03.y += reg_a4 * vec_b0_3.y; vec_c4_03.z += reg_a4 * vec_b0_3.z; vec_c4_03.w += reg_a4 * vec_b0_3.w;
            vec_c4_47.x += reg_a4 * vec_b4_7.x; vec_c4_47.y += reg_a4 * vec_b4_7.y; vec_c4_47.z += reg_a4 * vec_b4_7.z; vec_c4_47.w += reg_a4 * vec_b4_7.w;

            vec_c5_03.x += reg_a5 * vec_b0_3.x; vec_c5_03.y += reg_a5 * vec_b0_3.y; vec_c5_03.z += reg_a5 * vec_b0_3.z; vec_c5_03.w += reg_a5 * vec_b0_3.w;
            vec_c5_47.x += reg_a5 * vec_b4_7.x; vec_c5_47.y += reg_a5 * vec_b4_7.y; vec_c5_47.z += reg_a5 * vec_b4_7.z; vec_c5_47.w += reg_a5 * vec_b4_7.w;

            vec_c6_03.x += reg_a6 * vec_b0_3.x; vec_c6_03.y += reg_a6 * vec_b0_3.y; vec_c6_03.z += reg_a6 * vec_b0_3.z; vec_c6_03.w += reg_a6 * vec_b0_3.w;
            vec_c6_47.x += reg_a6 * vec_b4_7.x; vec_c6_47.y += reg_a6 * vec_b4_7.y; vec_c6_47.z += reg_a6 * vec_b4_7.z; vec_c6_47.w += reg_a6 * vec_b4_7.w;

            vec_c7_03.x += reg_a7 * vec_b0_3.x; vec_c7_03.y += reg_a7 * vec_b0_3.y; vec_c7_03.z += reg_a7 * vec_b0_3.z; vec_c7_03.w += reg_a7 * vec_b0_3.w;
            vec_c7_47.x += reg_a7 * vec_b4_7.x; vec_c7_47.y += reg_a7 * vec_b4_7.y; vec_c7_47.z += reg_a7 * vec_b4_7.z; vec_c7_47.w += reg_a7 * vec_b4_7.w;


        }
        __syncthreads();
    }
    *reinterpret_cast<float4*>(&d_C(Ci + 0, Cj + 0)) = vec_c0_03; *reinterpret_cast<float4*>(&d_C(Ci + 0, Cj + 4)) = vec_c0_47;
    *reinterpret_cast<float4*>(&d_C(Ci + 1, Cj + 0)) = vec_c1_03; *reinterpret_cast<float4*>(&d_C(Ci + 1, Cj + 4)) = vec_c1_47;
    *reinterpret_cast<float4*>(&d_C(Ci + 2, Cj + 0)) = vec_c2_03; *reinterpret_cast<float4*>(&d_C(Ci + 2, Cj + 4)) = vec_c2_47;
    *reinterpret_cast<float4*>(&d_C(Ci + 3, Cj + 0)) = vec_c3_03; *reinterpret_cast<float4*>(&d_C(Ci + 3, Cj + 4)) = vec_c3_47;
    *reinterpret_cast<float4*>(&d_C(Ci + 4, Cj + 0)) = vec_c4_03; *reinterpret_cast<float4*>(&d_C(Ci + 4, Cj + 4)) = vec_c4_47;
    *reinterpret_cast<float4*>(&d_C(Ci + 5, Cj + 0)) = vec_c5_03; *reinterpret_cast<float4*>(&d_C(Ci + 5, Cj + 4)) = vec_c5_47;
    *reinterpret_cast<float4*>(&d_C(Ci + 6, Cj + 0)) = vec_c6_03; *reinterpret_cast<float4*>(&d_C(Ci + 6, Cj + 4)) = vec_c6_47;
    *reinterpret_cast<float4*>(&d_C(Ci + 7, Cj + 0)) = vec_c7_03; *reinterpret_cast<float4*>(&d_C(Ci + 7, Cj + 4)) = vec_c7_47;
}


float sgemm_fast(int k, int m, int n,
    float* A, int lda,
    float* B, int ldb,
    float* C, int ldc)
{
    float* d_A, * d_B, * d_C;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&d_A, sizeof(float) * m * k);
    hipMalloc(&d_B, sizeof(float) * k * n);
    hipMalloc(&d_C, sizeof(float) * m * n);
    hipMemcpy(d_A, A, sizeof(float) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, sizeof(float) * k * n, hipMemcpyHostToDevice);
    hipMemset(d_C, 0, sizeof(float) * m * n);

    dim3 dim_block((m - 1) / (KERNEL_SIZE * REG_TILE_SIZE) + 1, (n - 1) / (KERNEL_SIZE * REG_TILE_SIZE) + 1, 1),
        dim_thread(KERNEL_SIZE, KERNEL_SIZE, 1);

    hipEventRecord(start, 0);
    sgemm_fast_kernel_optimiz_6 << <dim_block, dim_thread >> > (k, m, n,
        d_A, d_B, d_C);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    hipMemcpy(C, d_C, sizeof(float) * m * n, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return elapsedTime;
}